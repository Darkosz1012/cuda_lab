

hipError_t err;
err = hipMallocManaged(&a, N)                    // Assume the existence of `a` and `N`.

if (err != hipSuccess)                           // `hipSuccess` is provided by CUDA.
{
  printf("Error: %s\n", hipGetErrorString(err)); // `hipGetErrorString` is provided by CUDA.
}

/*
 * This launch should cause an error, but the kernel itself
 * cannot return it.
 */

someKernel<<<1, -1>>>();  // -1 is not a valid number of threads.

hipError_t err;
err = hipGetLastError(); // `hipGetLastError` will return the error from above.
if (err != hipSuccess)
{
  printf("Error: %s\n", hipGetErrorString(err));
}

// error wrapper
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}


