#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <fstream>
#include <chrono>
#include <iostream>



__global__ void matrixMultiplication2D(const float *A, const float *B, float *C, int size) {
	
	int rowIdx = blockIdx.y * size + threadIdx.y;
	int colIdx = blockIdx.x * size + threadIdx.x;
	
	if(rowIdx < size && colIdx < size){
		
		float product = 0;

		for(int i = 0; i < size; i++){

			product += A[rowIdx * size + i] * B[i * size + colIddx];
		}
		
		C[rowIdx * size + colIdx] = product;
	}
}

__global__ void matrixMultiplication3D(const float *A, const float *B, float *C, int size) {

	int rowIdx = blockIdx.y * size + threadIdx.y;
	int colIdx = blockIdx.x * size + threadIdx.x;
	int deepIdx = blockIdx.x * size + threadIdx.x;
	if(rowIdx < size && colIdx < size && deepIdx < size){
		C[rowIdx * size + colIdx] += A[rowIdx * size + deepIdx] * B[deepIdx * size + colIddx];
	}
}

void checkMatrixMul( int * a, int * b, int * c )
{
    int val = 0;

    for( int row = 0; row < N; ++row )
        for( int col = 0; col < N; ++col )
        {
            val = 0;
            for ( int k = 0; k < N; ++k )
                val += a[row * N + k] * b[k * N + col];
            if(c[row * N + col] != val)
                std::cout<<"Error: Result"<<std::endl;
        }
}

inline hipError_t checkCUDA(hipError_t result){

	if(result != hipSuccess){
	
	fprintf(stderr, "CUDA Runtime error: %s\n", hipGetErrorString(result));
	assert(result == hipSuccess);
	}	
	
	return result;
}

void allocWithCPU(float* h_A, float* h_B, float* h_result, float* d_A, float* d_B, float* d_result,size_t size,int numberOfElemets){
    //classic mallocs
    h_A = static_cast<float*>(malloc(size));
    h_B = static_cast<float*>(malloc(size));
    h_result = static_cast<float*>(malloc(size));

    for(int j = 0; j < numberOfElements; j++){
        h_A[j] = static_cast<float>(rand())/RAND_MAX;
        h_B[j] = static_cast<float>(rand())/RAND_MAX;
        h_result[j]=0;
    }

    checkCUDA(hipMalloc((void**)&d_A, size));
    checkCUDA(hipMalloc((void**)&d_B, size));
    checkCUDA(hipMalloc((void**)&d_result, size));

    checkCUDA(hipMemcpy(d_A, h_A, hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(d_B, h_B, hipMemcpyHostToDevice));
}
void allocWithGPU(float* A, float* B, float* result,size_t size,int numberOfElemets){
    checkCUDA(hipMallocManaged(&A, size));
    checkCUDA(hipMallocManaged(&B, size));
    checkCUDA(hipMallocManaged(&result, size));

    for(int j = 0; j < numberOfElements; j++){
        A[j] = static_cast<float>(rand())/RAND_MAX;
        B[j] = static_cast<float>(rand())/RAND_MAX;
        result[j] = 0;
    }

}
int main() {
	
	float* h_A, h_B, h_result;
	float* d_A, d_B, d_result;

	float* A, B, result;

	int numberOfElementsInDim = 10;
	int numberOfElemets = numberOElementsInDim*NumberOfElementsInDim;
	size_t size = numberOfElements * sizeof(float);

	std::ofstream save;

	std::chrono::high_resolution_clock start;
	std::chrono::high_resolution_clock stop;
	std::chronoduration<double> elapsed_time;

    int jump = 100;
    int numberOfResult = 100;

    save.open("classic_2D.txt");
	for(int i = 0; i < numberOfResult; i++){
        double average = 0;
		for(int k = 0; k < 10; k++){
			start = std::chrono::high_resolution_clock::now();

			allocWithCPU(h_A, h_B, h_result, d_A, d_B, d_result, size, numberOfElements);

            dim3 threads_per_block (16, 16, 1); // A 16 x 16 block threads
            dim3 number_of_blocks ((numberOfElementsInDim / threads_per_block.x) + 1, (numberOfElementsInDim / threads_per_block.y) + 1, 1);
			matrixMultiplication2D<<<number_of_blocks, threads_per_block>>>(d_A, d_B, d_result, numberOfElementsInDim);

            hipDeviceSynchronize();

			stop = std::chrono::high_resolution_clock::now();
			elapsed_time = stop - start;
            average = (average*k+elapsed_time)/k+1;

            checkCUDA(hipMemcpy(d_result, h_result, hipMemcpyDeviceToHost));
            
            checkMatrixMul(h_A,h_B,h_result);

			checkCUDA(hipFree(d_A));
			checkCUDA(hipFree(d_B));
			checkCUDA(hipFree(d_result));
			free(h_A);
			free(h_B);
			free(h_result);

		}
        save << numberOfElements <<"\t"<< numberOfElemetnsInDim <<"\t" << average << std::endl;
		
        
		numberOfElemetnsInDim += jump;
		numberOfElements = numberOfElemetnsInDim * numberOfElemetnsInDim;
		size = numberOfElemetns * sizeof(float);
	}
    save.close();

    save.open("managed_2D.txt");
	for(int i = 0; i < numberOfResult; i++){
        double average = 0;
		for(int k = 0; k < 10; k++){
			start = std::chrono::high_resolution_clock::now();

			allocWithGPU(A, B, result, size, numberOfElements);

            dim3 threads_per_block (16, 16, 1); // A 16 x 16 block threads
            dim3 number_of_blocks ((numberOfElementsInDim / threads_per_block.x) + 1, (numberOfElementsInDim / threads_per_block.y) + 1, 1);
			matrixMultiplication2D<<<number_of_blocks, threads_per_block>>>(A, B, result, numberOfElementsInDim);

            hipDeviceSynchronize();

			stop = std::chrono::high_resolution_clock::now();
			elapsed_time = stop - start;
            average = (average*k+elapsed_time)/k+1;
            
            checkMatrixMul(A,B,result);

            checkCUDA(hipFree(A));
            checkCUDA(hipFree(B));
            checkCUDA(hipFree(result));

		}
        save << numberOfElements <<"\t"<< numberOfElemetnsInDim <<"\t" << average << std::endl;
		
        
		numberOfElemetnsInDim += jump;
		numberOfElements = numberOfElemetnsInDim * numberOfElemetnsInDim;
		size = numberOfElemetns * sizeof(float);
	}
    save.close();

   

	return 0;
}

