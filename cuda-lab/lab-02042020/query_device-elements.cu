
/// recommended query to the device

  int deviceId;
  hipGetDevice(&deviceId);

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  /*
   * `props` now contains several properties about the current device.
   */

  int computeCapabilityMajor = props.major;
  int computeCapabilityMinor = props.minor;
  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;


