#include "hip/hip_runtime.h"
// example of using CUDA streams

#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;
	float* d;
	float* e;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
	hipMallocManaged(&d, size);
	hipMallocManaged(&e, size);


  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);
	hipMemPrefetchAsync(d, size, deviceId);
	hipMemPrefetchAsync(e, size, deviceId);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  /*
   * Create 3 streams to run initialize the 3 data vectors in parallel.
   */

  hipStream_t stream1, stream2, stream3;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);

  /*
   * Give each `initWith` launch its own non-standard stream.
   */

  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, c, N);
  //initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
  //initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
  //initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);
  
	
  //addVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(c, a, b, N/3);
  //addVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(c+N/3,a+N/3, b+N/3, N/3 + 1);
  //addVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(c+2*N/3, a+2*N/3, b+2*N/3, N/3 + 1); 
	addVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(c, a, b, N);
	addVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(d, a, b, N);
	addVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(e, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  hipMemPrefetchAsync(c, size, hipCpuDeviceId);

  checkElementsAre(7, c, N);

  /*
   * Destroy streams when they are no longer needed.
   */

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}

