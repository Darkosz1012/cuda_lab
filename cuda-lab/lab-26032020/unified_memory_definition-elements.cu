// CPU-only

int N = 2<<20;
size_t size = N * sizeof(int);

int *a;
a = (int *)malloc(size);

// Use `a` in CPU-only program.

free(a);

// Accelerated

int N = 2<<20;
size_t size = N * sizeof(int);

int *a;
// Note the address of `a` is passed as first argument.
hipMallocManaged(&a, size);

// Use `a` on the CPU and/or on any GPU in the accelerated system.

hipFree(a);




